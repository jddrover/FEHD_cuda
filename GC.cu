#include "hip/hip_runtime.h"
#include "GC.h"
#include "dataContainers.h"
#include "utility.h"
#include "timeSeriesOPs.h"
#include <vector>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "kernels.h"
#include <complex>
#include <chrono>
#include <cblas.h>
#include <algorithm>
#include "mkARGPU.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "workArray.h"

void granger(std::vector<float> angleArray,
	     std::vector<float> &GCvals, paramContainer params,
	     int numComps,workForGranger workArray)
{
  int blksize = 1024;
  int grdsize = (int)(params.numParticles+blksize-1)/blksize;
  const dim3 blockSize(blksize);
  const dim3 gridSize(grdsize);

  float2 alphaC;
  float2 betaC;
  alphaC.x=1.0f;
  alphaC.y=0.0f;
  betaC.x=0.0f;
  betaC.y=0.0f;

  float2 alphaC2;
  float2 betaC2;
  alphaC2.x=-1.0f;
  alphaC2.y=0.0f;
  betaC2.x=1.0f;
  betaC2.y=0.0f;
  
  int lwork = workArray.lworkVal;

  
  
  // cublas handle. Contains info about the system that routines need.
  hipblasHandle_t cublasH = 0;
  hipblasCreate(&cublasH);

  // cusolver options - don't sort, don't compute eigenvectors, use
  // the upper triangle the matrix is Hermitian
  const int sort_eig = 0;
  const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
  const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

  hipsolverSyevjInfo_t syevj_params = NULL;
  
  hipsolverHandle_t cusolverH = NULL;
  hipsolverDnCreate(&cusolverH);

  hipsolverDnCreateSyevjInfo(&syevj_params);
  hipsolverDnXsyevjSetSortEig(syevj_params, sort_eig);
  // Multiply A'*Q which will create the M*L x M*P matrix
  // [A_1TQ1T  ... A_1TQPT]
  // [...                 ]
  // [A_LTQ2T  ... A_LTQPT]

  // Recall Q was stored as transposed matrices, so we use the
  // following gemm call to do all particles and lags at once.

  //const float alpha=1.0;
  //const float beta=0.0;

  int grdsize2 =
  (int)(numComps*numComps*params.numLags*params.numParticles+blksize-1)/blksize; 

  const dim3 gridSize2(grdsize2);

  int grdsize3 =
  (int)(numComps*numComps*params.numParticles*params.numFreqs+blksize-1)/blksize;

  const dim3 blockSizeTF(blksize);
  const dim3 gridSizeTF(grdsize3);
  //const int memsizetf = sizeof(float2)*blksize;
  float dt = 1.0f/(float)(params.sampRate);

  int grdsize4 =
  (int)((numComps*params.numFreqs*params.numParticles+blksize-1)/blksize);

  const dim3 gridSizeScale(grdsize4);


  int grdsize5 =
  (int)(params.numParticles*params.numFreqs*(numComps-1)*(numComps-1)+blksize-1)/blksize;

  const dim3 blockSizeShrink(blksize);
  const dim3 gridSizeShrink(grdsize5);

  int grdsize6 = (int)(params.numParticles*params.numFreqs+blksize-1)/blksize;
  const dim3 blockSizeProd(blksize);
  const dim3 gridSizeProd(grdsize6);
  
  const int memsizeEig = sizeof(float)*blksize;


  int grdsize7 = (int)(params.numParticles+blksize-1)/blksize;
  const dim3 blockSize_det2GC(blksize);
  const dim3 gridSize_det2GC(grdsize7);



  // I remember this being here because it was difficult otherwise, I do not remember
  // what made it difficult. I would like to place it externally. 3/3/25
  // Create an array and allocate space on the device to store the rotation matrices
  float *angles_dev;
  hipMalloc((void**)&angles_dev, sizeof(float)*(numComps-1)*params.numParticles);

  hipMemcpy(angles_dev,angleArray.data(),sizeof(float)*(numComps-1)*params.numParticles,
	     hipMemcpyHostToDevice);


  // Using the angles in angles_dev, create the rotation matrices Q.
  generateRotationMatrices<<<gridSize,blockSize>>>(angles_dev,workArray.Qdev,numComps,params.numParticles);
  // This might have given the transpose - check
  // Outputs [Q1*,....,Qp*]
  // Create
  // workArray.AR holds the sums for each frequency
  // Need the Q's to be complex capable.

  hipComplex cu_alpha = make_hipComplex(1.0,0.0);
  hipComplex cu_beta = make_hipComplex(0.0,0.0);
  hipblasCgemm(cublasH,HIPBLAS_OP_C,HIPBLAS_OP_N,
	      params.numLags*numComps,params.numParticles*numComps,numComps,
	      &cu_alpha,workArray.ARdev,numComps,
	      workArray.Qdev,numComps,
	      &cu_beta,workArray.rotatedModels,params.numLags*numComps);

  transposeBlockMatrices<<<gridSize2,blockSize>>>(workArray.rotatedModels,workArray.wArray,numComps,params.numParticles,params.numFreqs);

  hipblasCgemmStridedBatched(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
			    params.numFreqs*numComps,numComps,numComps,
			    &cu_alpha,workArray.wArray,params.numFreqs*numComps,params.numFreqs*numComps*numComps,
			    workArray.Qdev,numComps,numComps*numComps,
			    &cu_beta,workArray.rotatedModels,numComps*params.numFreqs,params.numFreqs*numComps*numComps,
			    params.numParticles);

  // Form this into the long array.
  blksize = 1024;
  grdsize = (int)(params.numParticles*params.numFreqs*numComps*numComps+blksize-1)/blksize;
  const dim3 blockReform(blksize);
  const dim3 gridReform(grdsize);
  
  reformat<<<gridReform,blockReform>>>(workArray.rotatedModels,workArray.Tf,params,numComps);
  

  
  

  // Compute (Tf Tf*)^-1=Tf*^-1 Tf^-1 - the inverse of the variance in the neighborhood of each frequency
  // Collectively the inverse power spectrum of the model. 
  hipblasCgemmStridedBatched(cublasH,HIPBLAS_OP_C,HIPBLAS_OP_N,
				   numComps,numComps,numComps,
				   &alphaC,
				   workArray.Tf,numComps,numComps*numComps,
				   workArray.Tf,numComps,numComps*numComps,
				   &betaC,
				   workArray.Swhole,numComps,numComps*numComps,
				   params.numParticles*params.numFreqs);

  // Our first goal is to obtain the sub m-1 x m-1 spectral matrix for each frequency.
  // We have avoided inversion, so we don't have the spectral matrix, we have its inverse.
  // Consider the block matrix inverse:
  // [A  B]^-1   [[A-CB/D]^-1 X]
  // [    ]    = [             ]
  // [C  D]      [ X          X]
  // This is the inverse of the spectral sub matrix. We're going to calculate its determinant, so
  // we will not need to invert it.
  // This function scales the mth column (B) in each submatrix by the m,m entry.
  // This is preparation for the gemm below. 
  scale_columns<<<gridSizeScale,blockSizeTF>>>(workArray.Swhole,numComps,params.numParticles,params.numFreqs);
  // Copy the entire spectral matrix to a temporary array (not really temporary)
  hipblasCcopy(cublasH,params.numParticles*params.numFreqs*numComps*numComps,
	      workArray.Swhole,1,workArray.tmp,1);
  // GEMM does the above calculation.
  hipblasCgemmStridedBatched(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
				   numComps-1,numComps-1,1,
				   &alphaC2,
				   workArray.Swhole+(numComps-1)*numComps,numComps,numComps*numComps,
				   workArray.Swhole+(numComps-1),numComps,numComps*numComps, 
				   &betaC2,
				   workArray.tmp,numComps,numComps*numComps,
				   params.numParticles*params.numFreqs);
  // Copy back to Swhole array
  hipblasCcopy(cublasH,params.numParticles*params.numFreqs*numComps*numComps,
	      workArray.tmp,1,workArray.Swhole,1);

  // Same trick, but we need the product of the sub-transfer functions.
  // We determine the sub-inverse as above, first by scaling:
  scale_columns<<<gridSizeScale,blockSizeTF>>>(workArray.Tf,numComps,params.numParticles,params.numFreqs);
  // Then copying to a temporary array
  hipblasCcopy(cublasH,params.numParticles*params.numFreqs*numComps*numComps,
	      workArray.Tf,1,workArray.tmp,1);
  // Now we have the inverses of the sub transfer functions.
  hipblasCgemmStridedBatched(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
				   numComps-1,numComps-1,1,
				   &alphaC2,
				   workArray.Tf+(numComps-1)*numComps,numComps,numComps*numComps,
				   workArray.Tf+numComps-1,numComps,numComps*numComps,
				   &betaC2,
				   workArray.tmp,numComps,numComps*numComps,
				   params.numParticles*params.numFreqs);

  // And we multiply them to get the spectrum without the influence of the last component.
  hipblasCgemmStridedBatched(cublasH,HIPBLAS_OP_C,HIPBLAS_OP_N,
				   numComps-1,numComps-1,numComps-1,
				   &alphaC,
				   workArray.tmp,numComps,numComps*numComps,
				   workArray.tmp,numComps,numComps*numComps,
				   &betaC,
				   workArray.Spartial,numComps-1,(numComps-1)*(numComps-1),
				   params.numParticles*params.numFreqs);
  // Cheev batched doesn't stride, so I shrink the whole spectrum arrays to the m-1 x m-1 size. 
  shrinkArrays<<<gridSizeShrink,blockSizeShrink>>>(workArray.Swhole, workArray.d_wholeSpec, numComps, params.numParticles, params.numFreqs);
  // Cholesky algorithm to determine the eigenvalues (we set it not to compute eigenvectors, it can)
  hipsolverDnCheevjBatched(cusolverH,jobz,uplo,numComps-1,workArray.d_wholeSpec,numComps-1,
			  workArray.dev_W, workArray.d_work2,lwork,workArray.d_info, syevj_params, params.numFreqs*params.numParticles);
  // Multiply the eigenvalues together to get the determinant. 
  prodEigs<<<gridSizeProd,blockSizeProd,memsizeEig>>>(workArray.dev_W, workArray.det_whole, numComps-1, params.numParticles, params.numFreqs);
  // Repeat for the partial spectral matrices. 
  hipsolverDnCheevjBatched(cusolverH,jobz,uplo,numComps-1,workArray.Spartial,numComps-1,
			  workArray.dev_W, workArray.d_work2,lwork,workArray.d_info, syevj_params, params.numFreqs*params.numParticles);
  // Compute the determinant.
  prodEigs<<<gridSizeProd,blockSizeProd,memsizeEig>>>(workArray.dev_W, workArray.det_partial, numComps-1, params.numParticles, params.numFreqs);      
  // Divides the determinants, takes the log, and adds to the integral. 
  det2GC<<<gridSize_det2GC,blockSize_det2GC>>>(workArray.det_partial, workArray.det_whole, workArray.dev_GC,params.numParticles,params.numFreqs);
  // Send the numParticles Granger causality values to the system memory.
  hipMemcpy(GCvals.data(),workArray.dev_GC,sizeof(float)*params.numParticles,hipMemcpyDeviceToHost);
  // Clean up (if you don't memory will leak).
  hipsolverDnDestroy(cusolverH);
  hipblasDestroy(cublasH);
  hipFree(angles_dev);

  
  return;
}

void runFEHDstep(std::vector<float> &bestAngle, matrix &L, dataList dataArray ,paramContainer params,int numComps)
{

  // Determine the available memory on the GPU
  int id;
  size_t freemem,total;
  hipGetDevice(&id);
 
  srand((unsigned)time(0));

  ARmodel A;
  dataList residuals;
  std::vector<int> lagList(params.lagList);

  std::sort(lagList.begin(),lagList.end());

  mkARGPU(dataArray, lagList, A, residuals);
  
  // Orthonormalize the residuals using the SVD.
  dataList ortho_residuals;

  // Obtain the transformation that orthonormalizes the residual time series.
  orthonormalizeR(residuals, ortho_residuals, L); // This function is in mkARGPU.h
  // Apply the transformations - LAL^-1
  rotate_model(A, L); // Also in mkARGPU.h
  /*
  // Convert the AR model format to a single vector so it can be copied etc.
  std::vector<std::complex<float>> AR(params.numLags*numComps*numComps,std::complex<float>(0.0,0.0));
  for(int lag=0;lag<params.numLags;lag++)
    for(int row=0;row<numComps;row++)
      for(int col=0;col<numComps;col++)
	{
	  AR[lag*numComps*numComps+col*numComps+row] = std::complex<float>(A.lagMatrices[lag].elements[col*numComps+row],0.0);
	}

  std::vector<std::complex<float>> Tfhost(params.numFreqs*numComps*numComps,std::complex<float>(0.0,0.0));
  hipComplex alfa;
  std::vector<float> freq(params.numFreqs,0.0);

  
  // One particle, identity rotation.
  for(int findx=0;findx<params.numFreqs;findx++)
    {
      freq[findx] = (params.freqHi-params.freqLo)/(params.numFreqs-1)*float(findx)+params.freqLo;
      for(int lag=0;lag<params.numLags;lag++)
	{
	  alfa = make_hipComplex(-2.0*M_PI*freq[findx]*lagList[lag]/params.sampRate,0.0);
	  cblas_caxpy(numComps*numComps,&alfa,AR.data()+lag*numComps*numComps,1,Tfhost.data()+findx*numComps*numComps,1);
	}
    }				  
					  
  
  // The step-sizes to check along the (-)gradient.
  std::vector<float> h = {0.001f, 0.01f, 0.1f};

  std::vector<float> candidates(4,0);
  int minIndx;
  float allBlockMin=10000.0; // Just needs a somewhat large value. Will be set below.

  int minBlockNumber;
  // For recycling
  unsigned long int minimumGC;
  unsigned long int allBlockParticle;
  std::vector<int> resetList;
  std::vector<float> angleArrayReset;

  std::vector<float> GCvalsReset;

  paramContainer paramsReset = params;

  // Determine how to break up the analysis so that it fits on the GPU.
  int numBlocks, particleBlockSize;
  hipMemGetInfo(&freemem, &total);

  if(params.verbose)
    {
      printf("memory free = %ld bytes \n",freemem);
      printf("total memory = %ld bytes \n",total);
    }
  
  computeBlocks(numBlocks,particleBlockSize,freemem,params,numComps);
  
  paramContainer paramsBLOCKED = params;
  paramsBLOCKED.numParticles = particleBlockSize;

  if(params.verbose)
    {
      printf("Number of blocks = %i \n",numBlocks);
      printf("Block size = %i \n",particleBlockSize);
    }

  std::vector<float> GCmin(numBlocks,0);
  std::vector<int> GCminIndex(numBlocks,0);

  // Allocate all of the arrays need for the GC function. 
  workForGranger workArray;
  allocateParams(workArray,numComps,particleBlockSize,params,lagList,Tfhost);
   
  // Angle arrays.
  std::vector<std::vector<float>> angleArray;
  std::vector<std::vector<float>> angleArray1;
  std::vector<std::vector<float>> angleArray2;
  std::vector<std::vector<float>> angleArray3;

  std::vector<float> tmpAngle;
  
  for(int block=0;block<numBlocks;block++)
    {
      for(int indx=0;indx<particleBlockSize*(numComps-1);indx++)
	tmpAngle.push_back((float)(rand()%314-157)/100.0f);
  
      angleArray.push_back(tmpAngle);
      angleArray1.push_back(tmpAngle);
      angleArray2.push_back(tmpAngle);
      angleArray3.push_back(tmpAngle);
			    
      tmpAngle.clear();
    }

  // GCvals arrays - these store the Granger causality and
  // are the value we wish to minimize.
  std::vector<std::vector<float>> GCvals;
  std::vector<std::vector<float>> GCvals1;
  std::vector<std::vector<float>> GCvals2;
  std::vector<std::vector<float>> GCvals3;
  
  std::vector<float> GCtmp(particleBlockSize,0);
  
  for(int block=0;block<numBlocks;block++)
    {
      GCvals.push_back(GCtmp);
      GCvals1.push_back(GCtmp);
      GCvals2.push_back(GCtmp);
      GCvals3.push_back(GCtmp);
    }

  // gradient arrays for each block.
  std::vector<std::vector<float>> gradient;
  std::vector<float> gradientTmp(particleBlockSize*(numComps-1),0);

  for(int block=0;block<numBlocks;block++)
    gradient.push_back(gradientTmp);
      
  for(int block=0;block<numBlocks;block++)
    {
      granger(angleArray[block],GCvals[block], paramsBLOCKED,numComps,workArray);
    }
  // Here is the iterator - adjustments occur here.
  // while STATIONARY_COUNT < COUNTMAX


  int STATIONARY_COUNT = 0;
  const int COUNTMAX = params.STUCKCOUNT;

  
  
  //for(int iter=0;iter<numIts;iter++)
  int iter = 0;
  while(STATIONARY_COUNT < COUNTMAX)
    {
      // Get a bunch of gradients
      for(int block=0;block<numBlocks;block++)
	compGradient(gradient[block],GCvals[block],angleArray[block],paramsBLOCKED,numComps,workArray);

      // Assign values to the angles accordning to the gradient.
      for(int block=0;block<numBlocks;block++)
	{
	  angleArray1[block]=angleArray[block];
	  angleArray2[block]=angleArray[block];
	  angleArray3[block]=angleArray[block];
	  cblas_saxpy(particleBlockSize*(numComps-1), -h[0], gradient[block].data(), 1, angleArray1[block].data(),1);
	  cblas_saxpy(particleBlockSize*(numComps-1), -h[1], gradient[block].data(), 1, angleArray2[block].data(),1);
	  cblas_saxpy(particleBlockSize*(numComps-1), -h[2], gradient[block].data(), 1, angleArray3[block].data(),1);
	}

      // Evaluate the minimization candidates.
      for(int block=0;block<numBlocks;block++)
	{

	  granger(angleArray1[block],GCvals1[block],paramsBLOCKED,numComps,workArray);
	  granger(angleArray2[block],GCvals2[block],paramsBLOCKED,numComps,workArray);
	  granger(angleArray3[block],GCvals3[block],paramsBLOCKED,numComps,workArray);

	}

      
      // Determine the minimum value its location for each of the blocks
      // Recycle the particles that are local minima.
      for(int block=0;block<numBlocks;block++)
	{

	  minimumGC = std::distance(GCvals[block].begin(),std::min_element(GCvals[block].begin(),GCvals[block].end()));

	  resetList.clear();
	  GCvalsReset.clear();
	  angleArrayReset.clear();
	  
	  for(int particle=0;particle<particleBlockSize;particle++)
	    {	  	  	  
	      candidates[0] = GCvals[block][particle];
	      candidates[1] = GCvals1[block][particle];
	      candidates[2] = GCvals2[block][particle];
	      candidates[3] = GCvals3[block][particle];
	      
	      minIndx = std::distance(candidates.begin(),min_element(candidates.begin(),candidates.end()));

	      if(minIndx == 0) // Recycle these
		if(minimumGC != particle)
		  {
		    resetList.push_back(particle); // Store the particle numbers to be reset.
		    GCvalsReset.push_back(0.0); // This just adjusts the size, used below.
		    for(int comp=0;comp<numComps-1;comp++) // Reset the angle array, and make a copy for the reset run.
		      {
		        angleArray[block][particle*(numComps-1)+comp] = (float)(rand()%314-157)/100.0f;
			angleArrayReset.push_back(angleArray[block][particle*(numComps-1)+comp]);
		      }
		  }
	    
	      if(minIndx == 1)
		{
		  GCvals[block][particle] = GCvals1[block][particle];
		  std::copy(angleArray1[block].data()+particle*(numComps-1),angleArray1[block].data()+particle*(numComps-1)+numComps-1,
			    angleArray[block].data()+particle*(numComps-1));
		}
	      if(minIndx == 2)
		{
		  GCvals[block][particle] = GCvals2[block][particle];
		  std::copy(angleArray2[block].data()+particle*(numComps-1),angleArray2[block].data()+particle*(numComps-1)+numComps-1,
			    angleArray[block].data()+particle*(numComps-1));
		}
	      if(minIndx == 3)
		{
		  GCvals[block][particle] = GCvals3[block][particle];
		  std::copy(angleArray3[block].data()+particle*(numComps-1),angleArray3[block].data()+particle*(numComps-1)+numComps-1,
			    angleArray[block].data()+particle*(numComps-1));
		}
	    }

	  GCminIndex[block]=std::min_element(GCvals[block].begin(),GCvals[block].end())-GCvals[block].begin();
	  GCmin[block]=GCvals[block][GCminIndex[block]];
	  
	  paramsReset.numParticles = GCvalsReset.size();
       
	  if(paramsReset.numParticles>=1)
	    {

	      granger(angleArrayReset,GCvalsReset,paramsReset,numComps,workArray);

	      for(int resetParticle=0;resetParticle<paramsReset.numParticles;resetParticle++)
		GCvals[block][resetList[resetParticle]]=GCvalsReset[resetParticle];
	    }
	}

      // Find the minimum over all of the blocks
      minBlockNumber = std::min_element(GCmin.begin(),GCmin.end())-GCmin.begin();
      if(allBlockMin <= GCmin[minBlockNumber])
	STATIONARY_COUNT++;
      else
	STATIONARY_COUNT = 0;
      
      allBlockMin = GCmin[minBlockNumber];
      allBlockParticle = minBlockNumber*particleBlockSize+GCminIndex[minBlockNumber];

      
      
      if(params.verbose)
      	printf("iteration = %i, particle = %li, value = %e, exit count = %i \n",
	       iter,allBlockParticle,allBlockMin,STATIONARY_COUNT);
      iter++;
    }
  
  // Return the best angle.

  long unsigned int indexVal = GCminIndex[minBlockNumber];

  //printf("%li \n",indexVal);

  std::copy(angleArray[minBlockNumber].data()+indexVal*(numComps-1),angleArray[minBlockNumber].data()+indexVal*(numComps-1)+numComps-1,bestAngle.begin());

  
  freeWorkArray(workArray);
  */
  return;
 
}
void compGradient(std::vector<float> &gradient ,std::vector<float> GCvalsBASE,std::vector<float> angleArray,paramContainer params, int numComps,
		  workForGranger workArray)
{
  const int numVars = numComps-1;
  const float  h_val = 0.001f; // This is for the gradient spacing.
  
  std::vector<float> angle(angleArray); // Copy this
  std::vector<float> GCvalsUTIL(params.numParticles,0);

  // These are all the same size, can I allocate the arrays here?
  // I will have to rename the file to .cu
  // Is passing device arrays done normally? Weird that I don't know this.
  
  for(int varIndex=0;varIndex<numVars;varIndex++)
    {
      
      for(int particle=0;particle<params.numParticles;particle++)
	{
	  angle[particle*numVars+varIndex] += h_val;
	}

      granger(angle,GCvalsUTIL,params,numComps,workArray);

      for(int particle=0;particle<params.numParticles;particle++)
	{
	  if(std::isnan(GCvalsUTIL[particle]))
	    {
	      printf("GC nan in gradient \n");
	      exit(0);
	    }
	  if(std::isnan(GCvalsBASE[particle]))
	    {
	      printf("It's in the base \n");
	      exit(0);
	    }
	}
      
      for(int particle=0;particle<params.numParticles;particle++)
	{
	  
	  gradient[particle*numVars+varIndex] = (GCvalsUTIL[particle]-
							GCvalsBASE[particle])/h_val;

	  if(std::isnan(gradient[particle*numVars+varIndex]))
	    {
	      printf("isnan after the calculation \n");
	      printf("GCvalsUTIL = %f, GCvalsBASE = %f \n",GCvalsUTIL[particle],GCvalsBASE[particle]);
	      exit(0);
	    }
	  
	  angle[particle*numVars+varIndex] -= h_val;
	  
	}      
    }  
}

void computeBlocks(int &numBlocks,int &particleBlockSize,size_t memval,paramContainer params,int numComps)
{
  int k;
  unsigned long int required=(unsigned long int)((unsigned long int)params.numParticles*
						 (unsigned long int)(sizeof(float)*
								     ((1+2*params.numLags)*numComps*numComps+ // Qdev and workArrays
								      2*params.numFreqs+1+params.numFreqs*(numComps-1)+numComps-1)+ // determinants, GCval,eigenvalues, anglearray
								     sizeof(float2)*
								     (3*params.numFreqs*numComps*numComps+ // Tf, Swhole, tmp
								      4*params.numFreqs*(numComps-1)*(numComps-1))+ // D_wholeSpec,Spartial,2x work array
								     sizeof(int)*params.numFreqs)+ // info array
						 (unsigned long int)(sizeof(int)*params.numLags+sizeof(float)*params.numLags*numComps*numComps));// laglist, ARmodel
  
  unsigned long int allowable = (unsigned long int)(memval);
  allowable = (unsigned long int)((double)allowable*0.9);

  if(params.verbose)
    {
      printf("memory requested = %lu \n",required);
      printf("memory available per block = %lu \n",allowable);
    }
  if(required <= allowable)
    {
      numBlocks = 1;
      particleBlockSize = params.numParticles;
    }
  else
    {
      k = ceil(log2f((float)((float)required/(float)allowable)));
      numBlocks = pow(2,k);
      particleBlockSize = ceil(params.numParticles/numBlocks);
    }

  return;
}


  
  
