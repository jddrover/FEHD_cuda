#include "hip/hip_runtime.h"
#include "GC.h"
#include "dataContainers.h"
#include "utility.h"
#include "timeSeriesOPs.h"
#include <vector>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "kernels.h"
#include <complex>
#include <chrono>
#include <cblas.h>
#include <algorithm>
#include "mkARGPU.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

void granger(float *ARdev, std::vector<float> angleArray,
	     std::vector<float> &GCvals, paramContainer params, int
	     numComps,int *lagList_DEVICE, float *Qdev,float
	     *rotatedModels,float *workArray,float2 *Tf,float2
	     *Swhole,float2 *tmp,float2 *Spartial,float2 *d_wholeSpec,
	     float *dev_W,int *d_info,int &lworkVal,float2
	     *d_work2,float *det_whole,float *det_partial,float
	     *dev_GC)
{
  int blksize = 1024;
  int grdsize = (int)(params.numParticles+blksize-1)/blksize;
  const dim3 blockSize(blksize);
  const dim3 gridSize(grdsize);

  float2 alphaC;
  float2 betaC;
  alphaC.x=1.0f;
  alphaC.y=0.0f;
  betaC.x=0.0f;
  betaC.y=0.0f;

  float2 alphaC2;
  float2 betaC2;
  alphaC2.x=-1.0f;
  alphaC2.y=0.0f;
  betaC2.x=1.0f;
  betaC2.y=0.0f;
  
  int lwork = lworkVal;

  
  
  // cublas handle. Contains info about the system that routines need.
  hipblasHandle_t cublasH = 0;
  hipblasCreate(&cublasH);

  // cusolver options - don't sort, don't compute eigenvectors, use
  // the upper triangle the matrix is Hermitian
  const int sort_eig = 0;
  const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_NOVECTOR;
  const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

  hipsolverSyevjInfo_t syevj_params = NULL;
  
  hipsolverHandle_t cusolverH = NULL;
  hipsolverDnCreate(&cusolverH);

  hipsolverDnCreateSyevjInfo(&syevj_params);
  hipsolverDnXsyevjSetSortEig(syevj_params, sort_eig);
  // Multiply A'*Q which will create the M*L x M*P matrix
  // [A_1TQ1T  ... A_1TQPT]
  // [...                 ]
  // [A_LTQ2T  ... A_LTQPT]

  // Recall Q was stored as transposed matrices, so we use the
  // following gemm call to do all particles and lags at once.

  const float alpha=1.0;
  const float beta=0.0;

  int grdsize2 =
  (int)(numComps*numComps*params.numLags*params.numParticles+blksize-1)/blksize; 

  const dim3 gridSize2(grdsize2);

  int grdsize3 =
  (int)(numComps*numComps*params.numParticles*params.numFreqs+blksize-1)/blksize;

  const dim3 blockSizeTF(blksize);
  const dim3 gridSizeTF(grdsize3);
  const int memsizetf = sizeof(float2)*blksize;
  float dt = 1.0f/(float)(params.sampRate);

  int grdsize4 =
  (int)((numComps*params.numFreqs*params.numParticles+blksize-1)/blksize);

  const dim3 gridSizeScale(grdsize4);


  int grdsize5 =
  (int)(params.numParticles*params.numFreqs*(numComps-1)*(numComps-1)+blksize-1)/blksize;

  const dim3 blockSizeShrink(blksize);
  const dim3 gridSizeShrink(grdsize5);

  int grdsize6 = (int)(params.numParticles*params.numFreqs+blksize-1)/blksize;
  const dim3 blockSizeProd(blksize);
  const dim3 gridSizeProd(grdsize6);
  
  const int memsizeEig = sizeof(float)*blksize;


  int grdsize7 = (int)(params.numParticles+blksize-1)/blksize;
  const dim3 blockSize_det2GC(blksize);
  const dim3 gridSize_det2GC(grdsize7);



  
  // Create an array and allocate space on the device to store the rotation matrices
  float *angles_dev;
  hipMalloc((void**)&angles_dev, sizeof(float)*(numComps-1)*params.numParticles);

  hipMemcpy(angles_dev,angleArray.data(),sizeof(float)*(numComps-1)*params.numParticles,
	     hipMemcpyHostToDevice);


  // Using the angles in angles_dev, create the rotation matrices Q.
  generateRotationMatrices<<<gridSize,blockSize>>>(angles_dev,Qdev,numComps,params.numParticles);
  // Create
  // [A1^tQ1* A1^tQ2* ... A1^tQp*]
  // [A2^tQ1* ...                ]
  // [...                        ]
  // [AL^tQ1* ...     ... AL^tQp*]
  hipblasSgemm(cublasH,HIPBLAS_OP_T,HIPBLAS_OP_N,numComps*params.numLags,numComps*params.numParticles,
  	      numComps,&alpha,ARdev,numComps,Qdev,numComps,&beta,rotatedModels,
	      numComps*params.numLags);
  // Transpose each individual lag matrix
  // [Q1A1 Q2A1 ... ... QpA1]
  // [Q1A2 ...              ]
  // [...                   ]
  // [Q1AL ...  ... ... QpAL]
  transposeBlockMatrices<<<gridSize2,blockSize>>>(rotatedModels,workArray,numComps,params.numParticles,params.numLags);
  // Multiply, strided
  // [Q1A1]     [Q2A1]    ... [QpA1]
  // [ ...]Q1*  [... ]Q2* ... [... ]Qp*
  // [Q1AL]     [Q2AL]    ... [QpAL]
  hipblasSgemmStridedBatched(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
			    params.numLags*numComps,numComps,numComps,
			    &alpha,
			    workArray,numComps*params.numLags,params.numLags*numComps*numComps,
			    Qdev,numComps,numComps*numComps,
			    &beta,
			    rotatedModels,numComps*params.numLags,params.numLags*numComps*numComps,
			    params.numParticles);
  // Compute the inverse of the transfer function - numParticles * numFreqs complex matrices
  // [Tfp1f1^-1, Tfp1f2^-1, ... , Tfp1fF^-1, Tfp2f1^-1, ... TfppfF^-1]
  // See the function in kernels.cu for the details on how it works.
  compTransferFunc<<<gridSizeTF,blockSizeTF,memsizetf>>>(rotatedModels,Tf,lagList_DEVICE,numComps,
						       params.numParticles,params.freqLo,
						       params.freqHi,params.numFreqs,
						       params.numLags,dt);
  // Compute (Tf Tf*)^-1=Tf*^-1 Tf^-1 - the inverse of the variance in the neighborhood of each frequency
  // Collectively the inverse power spectrum of the model. 
  hipblasCgemmStridedBatched(cublasH,HIPBLAS_OP_C,HIPBLAS_OP_N,
				   numComps,numComps,numComps,
				   &alphaC,
				   Tf,numComps,numComps*numComps,
				   Tf,numComps,numComps*numComps,
				   &betaC,
				   Swhole,numComps,numComps*numComps,
				   params.numParticles*params.numFreqs);

  // Our first goal is to obtain the sub m-1 x m-1 spectral matrix for each frequency.
  // We have avoided inversion, so we don't have the spectral matrix, we have its inverse.
  // Consider the block matrix inverse:
  // [A  B]^-1   [[A-CB/D]^-1 X]
  // [    ]    = [             ]
  // [C  D]      [ X          X]
  // This is the inverse of the spectral sub matrix. We're going to calculate its determinant, so
  // we will not need to invert it.
  // This function scales the mth column (B) in each submatrix by the m,m entry.
  // This is preparation for the gemm below. 
  scale_columns<<<gridSizeScale,blockSizeTF>>>(Swhole,numComps,params.numParticles,params.numFreqs);
  // Copy the entire spectral matrix to a temporary array (not really temporary)
  hipblasCcopy(cublasH,params.numParticles*params.numFreqs*numComps*numComps,
	      Swhole,1,tmp,1);
  // GEMM does the above calculation.
  hipblasCgemmStridedBatched(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
				   numComps-1,numComps-1,1,
				   &alphaC2,
				   Swhole+(numComps-1)*numComps,numComps,numComps*numComps,
				   Swhole+(numComps-1),numComps,numComps*numComps, 
				   &betaC2,
				   tmp,numComps,numComps*numComps,
				   params.numParticles*params.numFreqs);
  // Copy back to Swhole array
  hipblasCcopy(cublasH,params.numParticles*params.numFreqs*numComps*numComps,
	      tmp,1,Swhole,1);

  // Same trick, but we need the product of the sub-transfer functions.
  // We determine the sub-inverse as above, first by scaling:
  scale_columns<<<gridSizeScale,blockSizeTF>>>(Tf,numComps,params.numParticles,params.numFreqs);
  // Then copying to a temporary array
  hipblasCcopy(cublasH,params.numParticles*params.numFreqs*numComps*numComps,
	      Tf,1,tmp,1);
  // Now we have the inverses of the sub transfer functions.
  hipblasCgemmStridedBatched(cublasH,HIPBLAS_OP_N,HIPBLAS_OP_N,
				   numComps-1,numComps-1,1,
				   &alphaC2,
				   Tf+(numComps-1)*numComps,numComps,numComps*numComps,
				   Tf+numComps-1,numComps,numComps*numComps,
				   &betaC2,
				   tmp,numComps,numComps*numComps,
				   params.numParticles*params.numFreqs);

  // And we multiply them to get the spectrum without the influence of the last component.
  hipblasCgemmStridedBatched(cublasH,HIPBLAS_OP_C,HIPBLAS_OP_N,
				   numComps-1,numComps-1,numComps-1,
				   &alphaC,
				   tmp,numComps,numComps*numComps,
				   tmp,numComps,numComps*numComps,
				   &betaC,
				   Spartial,numComps-1,(numComps-1)*(numComps-1),
				   params.numParticles*params.numFreqs);
  // Cheev batched doesn't stride, so I shrink the whole spectrum arrays to the m-1 x m-1 size. 
  shrinkArrays<<<gridSizeShrink,blockSizeShrink>>>(Swhole, d_wholeSpec, numComps, params.numParticles, params.numFreqs);
  // Cholesky algorithm to determine the eigenvalues (we set it not to compute eigenvectors, it can)
  hipsolverDnCheevjBatched(cusolverH,jobz,uplo,numComps-1,d_wholeSpec,numComps-1,
			  dev_W, d_work2,lwork,d_info, syevj_params, params.numFreqs*params.numParticles);
  // Multiply the eigenvalues together to get the determinant. 
  prodEigs<<<gridSizeProd,blockSizeProd,memsizeEig>>>(dev_W, det_whole, numComps-1, params.numParticles, params.numFreqs);
  // Repeat for the partial spectral matrices. 
  hipsolverDnCheevjBatched(cusolverH,jobz,uplo,numComps-1,Spartial,numComps-1,
			  dev_W, d_work2,lwork,d_info, syevj_params, params.numFreqs*params.numParticles);
  // Compute the determinant.
  prodEigs<<<gridSizeProd,blockSizeProd,memsizeEig>>>(dev_W, det_partial, numComps-1, params.numParticles, params.numFreqs);      
  // Divides the determinants, takes the log, and adds to the integral. 
  det2GC<<<gridSize_det2GC,blockSize_det2GC>>>(det_partial, det_whole, dev_GC,params.numParticles,params.numFreqs);
  // Send the numParticles Granger causality values to the system memory.
  hipMemcpy(GCvals.data(),dev_GC,sizeof(float)*params.numParticles,hipMemcpyDeviceToHost);
  // Clean up (if you don't memory will leak).
  hipsolverDnDestroy(cusolverH);
  hipblasDestroy(cublasH);
  hipFree(angles_dev);

  
  return;
}

void runFEHDstep(std::vector<float> &bestAngle, matrix &L, dataList dataArray ,paramContainer params,int numComps)
{

  // Determine the available memory on the GPU
  int id;
  size_t freemem,total;
  hipGetDevice(&id);
 
  srand((unsigned)time(0));

  ARmodel A;
  dataList residuals;
  std::vector<int> lagList(params.lagList);

  std::sort(lagList.begin(),lagList.end());

  mkARGPU(dataArray, lagList, A, residuals);
  
  // Orthonormalize the residuals using the SVD.
  dataList ortho_residuals;

  // Obtain the transformation that orthonormalizes the residual time series.
  orthonormalizeR(residuals, ortho_residuals, L); // This function is in mkARGPU.h
  // Apply the transformations - LAL^-1
  rotate_model(A, L); // Also in mkARGPU.h

  // Convert the AR model format to a single vector so it can be copied etc.
  std::vector<float> AR(params.numLags*numComps*numComps,0);
  for(int lag=0;lag<params.numLags;lag++)
    for(int row=0;row<numComps;row++)
      for(int col=0;col<numComps;col++)
	{
	  AR[lag*numComps*numComps+col*numComps+row] = A.lagMatrices[lag].elements[col*numComps+row];
	}


  // Number of iterations
  int numIts = 50;
  // The step-sizes to check along the (-)gradient.
  std::vector<float> h = {0.01f, 1.0f, 10.0f};

  std::vector<float> candidates(4,0);
  int minIndx;
  float allBlockMin;

  int minBlockNumber;
  // For recycling
  unsigned long int minimumGC;
  unsigned long int allBlockParticle;
  std::vector<int> resetList;
  std::vector<float> angleArrayReset;

  std::vector<float> GCvalsReset;

  paramContainer paramsReset = params;

  // Determine how to break up the analysis so that it fits on the GPU.
  int numBlocks, particleBlockSize;
  hipMemGetInfo(&freemem, &total);

  if(params.verbose)
    {
      printf("memory free = %ld bytes \n",freemem);
      printf("total memory = %ld bytes \n",total);
    }
  
  computeBlocks(numBlocks,particleBlockSize,freemem,params,numComps); // Need to write this.
  
  paramContainer paramsBLOCKED = params;
  paramsBLOCKED.numParticles = particleBlockSize;

  if(params.verbose)
    {
      printf("Number of blocks = %i \n",numBlocks);
      printf("Block size = %i \n",particleBlockSize);
    }

  std::vector<float> GCmin(numBlocks,0);
  std::vector<int> GCminIndex(numBlocks,0);
  // Allocate a whole bunch of stuff on the GPU.
  float *Qdev;
  hipMalloc((void**)&Qdev, sizeof(float)*particleBlockSize*numComps*numComps);
  float *rotatedModels;
  float *workArray;
  hipMalloc((void**)&rotatedModels, sizeof(float)*numComps*numComps*particleBlockSize*params.numLags);
  hipMalloc((void**)&workArray, sizeof(float)*numComps*numComps*particleBlockSize*params.numLags);
  float2 *Tf;
  hipMalloc((void**)&Tf,sizeof(float2)*numComps*numComps*particleBlockSize*params.numFreqs);
  float2 *Swhole;
  hipMalloc((void**)&Swhole,sizeof(float2)*numComps*numComps*params.numFreqs*particleBlockSize);
  float2 *tmp;
  hipMalloc(&tmp,sizeof(float2)*particleBlockSize*params.numFreqs*numComps*numComps);
  float2 *Spartial; // Make partial one size down
  hipMalloc((void**)&Spartial,sizeof(float2)*(numComps-1)*(numComps-1)*params.numFreqs*particleBlockSize);
  float2 *d_wholeSpec;
  hipMalloc(&d_wholeSpec,sizeof(float2)*params.numFreqs*particleBlockSize*(numComps-1)*(numComps-1));
  float *dev_W;
  hipMalloc(&dev_W,sizeof(float)*particleBlockSize*params.numFreqs*(numComps-1));
  int *d_info;
  hipMalloc(&d_info,sizeof(int)*particleBlockSize*params.numFreqs);
  int lworkVal = particleBlockSize*params.numFreqs*2*(numComps-1)*(numComps-1); 
  float2 *d_work2;
  hipMalloc(&d_work2,sizeof(float2)*lworkVal);
  float *det_whole;
  hipMalloc(&det_whole,sizeof(float)*particleBlockSize*params.numFreqs);
  float *det_partial;
  hipMalloc(&det_partial,sizeof(float)*particleBlockSize*params.numFreqs);
  float *dev_GC;
  hipMalloc(&dev_GC,sizeof(float)*particleBlockSize);
  int *lagList_DEVICE;
  hipMalloc(&lagList_DEVICE,sizeof(int)*lagList.size());
  hipMemcpy(lagList_DEVICE,lagList.data(),sizeof(int)*lagList.size(),hipMemcpyHostToDevice);
  float *ARdev;
  hipMalloc(&ARdev,sizeof(float)*AR.size());
  hipMemcpy(ARdev,AR.data(),sizeof(float)*AR.size(),hipMemcpyHostToDevice);

  // Angle arrays.
  std::vector<std::vector<float>> angleArray;
  std::vector<std::vector<float>> angleArray1;
  std::vector<std::vector<float>> angleArray2;
  std::vector<std::vector<float>> angleArray3;

  std::vector<float> tmpAngle;
  
  for(int block=0;block<numBlocks;block++)
    {
      for(int indx=0;indx<particleBlockSize*(numComps-1);indx++)
	tmpAngle.push_back((float)(rand()%314-157)/100.0f);
  
      angleArray.push_back(tmpAngle);
      angleArray1.push_back(tmpAngle);
      angleArray2.push_back(tmpAngle);
      angleArray3.push_back(tmpAngle);
			    
      tmpAngle.clear();
    }

  // GCvals arrays - these store the Granger causality and
  // are the value we wish to minimize.
  std::vector<std::vector<float>> GCvals;
  std::vector<std::vector<float>> GCvals1;
  std::vector<std::vector<float>> GCvals2;
  std::vector<std::vector<float>> GCvals3;
  
  std::vector<float> GCtmp(particleBlockSize,0);
  
  for(int block=0;block<numBlocks;block++)
    {
      GCvals.push_back(GCtmp);
      GCvals1.push_back(GCtmp);
      GCvals2.push_back(GCtmp);
      GCvals3.push_back(GCtmp);
    }

  // gradient arrays for each block.
  std::vector<std::vector<float>> gradient;
  std::vector<float> gradientTmp(particleBlockSize*(numComps-1),0);

  for(int block=0;block<numBlocks;block++)
    gradient.push_back(gradientTmp);
      
  for(int block=0;block<numBlocks;block++)
    {
      granger(ARdev,angleArray[block],GCvals[block], paramsBLOCKED,numComps,lagList_DEVICE,Qdev,rotatedModels,workArray,Tf,Swhole,tmp,
	      Spartial,d_wholeSpec,dev_W,d_info,lworkVal,d_work2,det_whole,det_partial,dev_GC);
    }
  // Here is the iterator - adjustments occur here.
  // while STATIONARY_COUNT < COUNTMAX
  for(int iter=0;iter<numIts;iter++)
    {
      // Get a bunch of gradients
      for(int block=0;block<numBlocks;block++)
	compGradient(ARdev,gradient[block],GCvals[block],angleArray[block],paramsBLOCKED,numComps,lagList_DEVICE,Qdev,rotatedModels,
		     workArray,Tf,Swhole,tmp,Spartial,d_wholeSpec,dev_W,d_info,lworkVal,d_work2,det_whole,det_partial,dev_GC);

      // Assign values to the angles accordning to the gradient.
      for(int block=0;block<numBlocks;block++)
	{
	  angleArray1[block]=angleArray[block];
	  angleArray2[block]=angleArray[block];
	  angleArray3[block]=angleArray[block];
	  cblas_saxpy(particleBlockSize*(numComps-1), -h[0], gradient[block].data(), 1, angleArray1[block].data(),1);
	  cblas_saxpy(particleBlockSize*(numComps-1), -h[1], gradient[block].data(), 1, angleArray2[block].data(),1);
	  cblas_saxpy(particleBlockSize*(numComps-1), -h[2], gradient[block].data(), 1, angleArray3[block].data(),1);
	}

      // Evaluate the minimization candidates.
      for(int block=0;block<numBlocks;block++)
	{
	  granger(ARdev,angleArray1[block],GCvals1[block],paramsBLOCKED,numComps,lagList_DEVICE,Qdev,rotatedModels,workArray,Tf,Swhole,tmp,Spartial,d_wholeSpec,dev_W,
		  d_info,lworkVal,d_work2,det_whole,det_partial,dev_GC);
	  granger(ARdev,angleArray2[block],GCvals2[block],paramsBLOCKED,numComps,lagList_DEVICE,Qdev,rotatedModels,workArray,Tf,Swhole,tmp,Spartial,d_wholeSpec,dev_W,
		  d_info,lworkVal,d_work2,det_whole,det_partial,dev_GC);
	  granger(ARdev,angleArray3[block],GCvals3[block],paramsBLOCKED,numComps,lagList_DEVICE,Qdev,rotatedModels,workArray,Tf,Swhole,tmp,Spartial,d_wholeSpec,dev_W,
		  d_info,lworkVal,d_work2,det_whole,det_partial,dev_GC);
	}

      
      // Determine the minimum value its location for each of the blocks
      // Recycle the particles that are local minima.
      for(int block=0;block<numBlocks;block++)
	{

	  minimumGC = std::distance(GCvals[block].begin(),std::min_element(GCvals[block].begin(),GCvals[block].end()));

	  resetList.clear();
	  GCvalsReset.clear();
	  angleArrayReset.clear();
	  
	  for(int particle=0;particle<particleBlockSize;particle++)
	    {	  	  	  
	      candidates[0] = GCvals[block][particle];
	      candidates[1] = GCvals1[block][particle];
	      candidates[2] = GCvals2[block][particle];
	      candidates[3] = GCvals3[block][particle];
	      
	      minIndx = std::distance(candidates.begin(),min_element(candidates.begin(),candidates.end()));

	      if(minIndx == 0) // Recycle these
		if(minimumGC != particle)
		  {
		    resetList.push_back(particle); // Store the particle numbers to be reset.
		    GCvalsReset.push_back(0.0); // This just adjusts the size, used below.
		    for(int comp=0;comp<numComps-1;comp++) // Reset the angle array, and make a copy for the reset run.
		      {
		        angleArray[block][particle*(numComps-1)+comp] = (float)(rand()%314-157)/100.0f;
			angleArrayReset.push_back(angleArray[block][particle*(numComps-1)+comp]);
		      }
		  }
	    
	      if(minIndx == 1)
		{
		  GCvals[block][particle] = GCvals1[block][particle];
		  std::copy(angleArray1[block].data()+particle*(numComps-1),angleArray1[block].data()+particle*(numComps-1)+numComps-1,
			    angleArray[block].data()+particle*(numComps-1));
		}
	      if(minIndx == 2)
		{
		  GCvals[block][particle] = GCvals2[block][particle];
		  std::copy(angleArray2[block].data()+particle*(numComps-1),angleArray2[block].data()+particle*(numComps-1)+numComps-1,
			    angleArray[block].data()+particle*(numComps-1));
		}
	      if(minIndx == 3)
		{
		  GCvals[block][particle] = GCvals3[block][particle];
		  std::copy(angleArray3[block].data()+particle*(numComps-1),angleArray3[block].data()+particle*(numComps-1)+numComps-1,
			    angleArray[block].data()+particle*(numComps-1));
		}
	    }

	  GCminIndex[block]=std::min_element(GCvals[block].begin(),GCvals[block].end())-GCvals[block].begin();
	  GCmin[block]=GCvals[block][GCminIndex[block]];
	  
	  paramsReset.numParticles = GCvalsReset.size();
       
	  if(paramsReset.numParticles>=1)
	    {
	      granger(ARdev,angleArrayReset,GCvalsReset,paramsReset,numComps,lagList_DEVICE,
		      Qdev,rotatedModels,workArray,Tf,Swhole,tmp,Spartial,d_wholeSpec,dev_W, d_info,
		      lworkVal,d_work2,det_whole,det_partial,dev_GC);
	      for(int resetParticle=0;resetParticle<paramsReset.numParticles;resetParticle++)
		GCvals[block][resetList[resetParticle]]=GCvalsReset[resetParticle];
	    }
	}

      // Find the minimum over all of the blocks
      minBlockNumber = std::min_element(GCmin.begin(),GCmin.end())-GCmin.begin();
      allBlockMin = GCmin[minBlockNumber];
      allBlockParticle = minBlockNumber*particleBlockSize+GCminIndex[minBlockNumber];

      
      
      if(params.verbose)
      	printf("iteration = %i, particle = %li, value = %e \n",
	       iter,allBlockParticle,allBlockMin);
  
    }
  
  // Return the best angle.

  long unsigned int indexVal = GCminIndex[minBlockNumber];

  //printf("%li \n",indexVal);

  std::copy(angleArray[minBlockNumber].data()+indexVal*(numComps-1),angleArray[minBlockNumber].data()+indexVal*(numComps-1)+numComps-1,bestAngle.begin());

  hipFree(Qdev);
  hipFree(rotatedModels);
  hipFree(workArray);
  hipFree(Tf);
  hipFree(Swhole);
  hipFree(tmp);
  hipFree(Spartial);
  hipFree(d_wholeSpec);
  hipFree(dev_W);
  hipFree(d_info);
  hipFree(d_work2);
  hipFree(det_whole);
  hipFree(det_partial);
  hipFree(dev_GC);
  hipFree(lagList_DEVICE);
  hipFree(ARdev);
  
  return;
 
}
void compGradient(float *ARdev, std::vector<float> &gradient ,std::vector<float> GCvalsBASE,std::vector<float> angleArray,paramContainer params, int numComps,
		  int *lagList_DEVICE,float *Qdev,float *rotatedModels,float *workArray,float2 *Tf,float2 *Swhole,float2 *tmp,float2 *Spartial,
		  float2 *d_wholeSpec,float *dev_W,int *d_info,int lworkVal,float2 *d_work2,float *det_whole,float *det_partial,float *dev_GC)
{
  const int numVars = numComps-1;
  const float  h_val = 0.001f; // This is for the gradient spacing.
  
  std::vector<float> angle(angleArray); // Copy this
  std::vector<float> GCvalsUTIL(params.numParticles,0);

  // These are all the same size, can I allocate the arrays here?
  // I will have to rename the file to .cu
  // Is passing device arrays done normally? Weird that I don't know this.
  
  for(int varIndex=0;varIndex<numVars;varIndex++)
    {
      
      for(int particle=0;particle<params.numParticles;particle++)
	{
	  angle[particle*numVars+varIndex] += h_val;
	}

      granger(ARdev,angle,GCvalsUTIL,params,numComps,lagList_DEVICE,Qdev,rotatedModels,workArray,Tf,Swhole,tmp,Spartial,d_wholeSpec,
	      dev_W,d_info,lworkVal,d_work2,det_whole,det_partial,dev_GC);

      for(int particle=0;particle<params.numParticles;particle++)
	{
	  if(std::isnan(GCvalsUTIL[particle]))
	    {
	      printf("GC nan in gradient \n");
	      exit(0);
	    }
	  if(std::isnan(GCvalsBASE[particle]))
	    {
	      printf("It's in the base \n");
	      exit(0);
	    }
	}
      
      for(int particle=0;particle<params.numParticles;particle++)
	{
	  
	  gradient[particle*numVars+varIndex] = (GCvalsUTIL[particle]-
							GCvalsBASE[particle])/h_val;

	  if(std::isnan(gradient[particle*numVars+varIndex]))
	    {
	      printf("isnan after the calculation \n");
	      printf("GCvalsUTIL = %f, GCvalsBASE = %f \n",GCvalsUTIL[particle],GCvalsBASE[particle]);
	      exit(0);
	    }
	  
	  angle[particle*numVars+varIndex] -= h_val;
	  
	}      
    }  
}

void computeBlocks(int &numBlocks,int &particleBlockSize,size_t memval,paramContainer params,int numComps)
{
  int k;
  unsigned long int required=(unsigned long int)((unsigned long int)params.numParticles*
						 (unsigned long int)(sizeof(float)*
								     ((1+2*params.numLags)*numComps*numComps+ // Qdev and workArrays
								      2*params.numFreqs+1+params.numFreqs*(numComps-1)+numComps-1)+ // determinants, GCval,eigenvalues, anglearray
								     sizeof(float2)*
								     (3*params.numFreqs*numComps*numComps+ // Tf, Swhole, tmp
								      4*params.numFreqs*(numComps-1)*(numComps-1))+ // D_wholeSpec,Spartial,2x work array
								     sizeof(int)*params.numFreqs)+ // info array
						 (unsigned long int)(sizeof(int)*params.numLags+sizeof(float)*params.numLags*numComps*numComps));// laglist, ARmodel
  
  unsigned long int allowable = (unsigned long int)(memval);
  allowable = (unsigned long int)((double)allowable*0.9);

  if(params.verbose)
    {
      printf("memory requested = %lu \n",required);
      printf("memory available per block = %lu \n",allowable);
    }
  if(required <= allowable)
    {
      numBlocks = 1;
      particleBlockSize = params.numParticles;
    }
  else
    {
      k = ceil(log2f((float)((float)required/(float)allowable)));
      numBlocks = pow(2,k);
      particleBlockSize = ceil(params.numParticles/numBlocks);
    }

  return;
}


  
  
