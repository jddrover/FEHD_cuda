#include "hip/hip_runtime.h"
#include "workArray.h"
#include "utility.h"
#include <vector>
void allocateParams(workForGranger &workArray,int numComps,int particleBlockSize,paramContainer params,std::vector<int> lagList,std::vector<float> AR)
{
  workArray.lworkVal = particleBlockSize*params.numFreqs*2*(numComps-1)*(numComps-1);
  hipMalloc((void**)&workArray.Qdev, sizeof(float)*particleBlockSize*numComps*numComps);
  hipMalloc((void**)&workArray.rotatedModels, sizeof(float)*numComps*numComps*particleBlockSize*params.numLags);
  hipMalloc((void**)&workArray.wArray, sizeof(float)*numComps*numComps*particleBlockSize*params.numLags);
  hipMalloc((void**)&workArray.Tf,sizeof(float2)*numComps*numComps*particleBlockSize*params.numFreqs);
  hipMalloc((void**)&workArray.Swhole,sizeof(float2)*numComps*numComps*params.numFreqs*particleBlockSize);
  hipMalloc(&workArray.tmp,sizeof(float2)*particleBlockSize*params.numFreqs*numComps*numComps);
  hipMalloc((void**)&workArray.Spartial,sizeof(float2)*(numComps-1)*(numComps-1)*params.numFreqs*particleBlockSize);
  hipMalloc(&workArray.d_wholeSpec,sizeof(float2)*params.numFreqs*particleBlockSize*(numComps-1)*(numComps-1));
  hipMalloc(&workArray.dev_W,sizeof(float)*particleBlockSize*params.numFreqs*(numComps-1));
  hipMalloc(&workArray.d_info,sizeof(int)*particleBlockSize*params.numFreqs);  
  hipMalloc(&workArray.d_work2,sizeof(float2)*workArray.lworkVal);
  hipMalloc(&workArray.det_whole,sizeof(float)*particleBlockSize*params.numFreqs);
  hipMalloc(&workArray.det_partial,sizeof(float)*particleBlockSize*params.numFreqs);
  hipMalloc(&workArray.dev_GC,sizeof(float)*particleBlockSize);
  hipMalloc(&workArray.lagList_DEVICE,sizeof(int)*lagList.size());
  hipMemcpy(workArray.lagList_DEVICE,lagList.data(),sizeof(int)*lagList.size(),hipMemcpyHostToDevice);
  hipMalloc(&workArray.ARdev,sizeof(float)*AR.size());
  hipMemcpy(workArray.ARdev,AR.data(),sizeof(float)*AR.size(),hipMemcpyHostToDevice);
  
  return;
}

void freeWorkArray(workForGranger &workArray)
{
  hipFree(workArray.Qdev);
  hipFree(workArray.rotatedModels);
  hipFree(workArray.wArray);
  hipFree(workArray.Tf);
  hipFree(workArray.Swhole);
  hipFree(workArray.tmp);
  hipFree(workArray.Spartial);
  hipFree(workArray.d_wholeSpec);
  hipFree(workArray.dev_W);
  hipFree(workArray.d_info);
  hipFree(workArray.d_work2);
  hipFree(workArray.det_whole);
  hipFree(workArray.det_partial);
  hipFree(workArray.dev_GC);
  hipFree(workArray.lagList_DEVICE);
  hipFree(workArray.ARdev);
  return;
}
